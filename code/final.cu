#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <algorithm>
#include <cstring>
#include <cstdlib>
#include <cfloat>

using namespace std;

// Global parameters
int MAX_ITER = 100000;
float THRESHOLD = 0.1f;

__device__ float calculateDistance(const float* point, const float* centroid, int dimensions) {
    float dist = 0;
    for (int i = 0; i < dimensions; ++i) {
        dist += (point[i] - centroid[i]) * (point[i] - centroid[i]);
    }
    return sqrtf(dist);
}

__global__ void assignCluster(const float* data, float* centroids, int* clusterAssignments, int numPoints, int dimensions, int clusters) {
    extern __shared__ float sharedCentroids[];
    int tid = threadIdx.x;

    for (int d = tid; d < clusters * dimensions; d += blockDim.x) {
        sharedCentroids[d] = centroids[d];
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPoints) {
        float minDist = FLT_MAX;
        int bestCluster = -1;

        for (int c = 0; c < clusters; ++c) {
            float dist = calculateDistance(&data[idx * dimensions], &sharedCentroids[c * dimensions], dimensions);
            if (dist < minDist) {
                minDist = dist;
                bestCluster = c;
            }
        }
        clusterAssignments[idx] = bestCluster;
    }
}

__global__ void updateCentroids(const float* data, const int* clusterAssignments, float* centroids, int* clusterCounts, int numPoints, int dimensions, int clusters) {
    extern __shared__ float sharedCentroids[];
    int tid = threadIdx.x;
    int clusterIdx = blockIdx.x;

    if (tid < dimensions) {
        sharedCentroids[tid] = 0.0f;
    }
    __syncthreads();

    for (int i = tid; i < numPoints; i += blockDim.x) {
        if (clusterAssignments[i] == clusterIdx) {
            for (int d = 0; d < dimensions; d++) {
                atomicAdd(&sharedCentroids[d], data[i * dimensions + d]);
            }
            atomicAdd(&clusterCounts[clusterIdx], 1);
        }
    }
    __syncthreads();

    if (tid < dimensions && clusterCounts[clusterIdx] > 0) {
        centroids[clusterIdx * dimensions + tid] = sharedCentroids[tid] / clusterCounts[clusterIdx];
    }
}

bool loadData(const string& filename, vector<float>& data, int& numPoints, int& dimensions, int& clusters) {
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error: Could not open file " << filename << endl;
        return false;
    }
    file >> numPoints >> dimensions >> clusters;
    float value;
    while (file >> value) {
        data.push_back(value);
    }
    file.close();
    return data.size() == numPoints * dimensions;
}

void initializeCentroids(const float* data, float* centroids, int numPoints, int dimensions, int clusters) {
    srand(42);
    for (int c = 0; c < clusters; ++c) {
        int idx = rand() % numPoints;
        for (int d = 0; d < dimensions; ++d) {
            centroids[c * dimensions + d] = data[idx * dimensions + d];
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        cerr << "Usage: " << argv[0] << " <threads_per_block>  <datasetFile>" << endl;
        return -1;
    }
    int threadsPerBlock = atoi(argv[1]);
    string datasetFile = argv[2];
    
    int numPoints, dimensions, clusters;
    vector<float> hostData;
    if (!loadData(datasetFile, hostData, numPoints, dimensions, clusters)) {
        return -1;
    }

    float *deviceData, *deviceCentroids;
    int *deviceAssignments, *deviceClusterCounts;
    hipMalloc(&deviceData, numPoints * dimensions * sizeof(float));
    hipMalloc(&deviceCentroids, clusters * dimensions * sizeof(float));
    hipMalloc(&deviceAssignments, numPoints * sizeof(int));
    hipMalloc(&deviceClusterCounts, clusters * sizeof(int));

    hipMemcpy(deviceData, hostData.data(), numPoints * dimensions * sizeof(float), hipMemcpyHostToDevice);

    vector<float> centroids(clusters * dimensions);
    initializeCentroids(hostData.data(), centroids.data(), numPoints, dimensions, clusters);
    hipMemcpy(deviceCentroids, centroids.data(), clusters * dimensions * sizeof(float), hipMemcpyHostToDevice);

    bool converged = false;
    int iteration = 0;

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    while (!converged && iteration < MAX_ITER) {
        iteration++;
        assignCluster<<<(numPoints + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock, clusters * dimensions * sizeof(float)>>>(deviceData, deviceCentroids, deviceAssignments, numPoints, dimensions, clusters);
        hipDeviceSynchronize();
        hipMemset(deviceClusterCounts, 0, clusters * sizeof(int));
        updateCentroids<<<clusters, threadsPerBlock, dimensions * sizeof(float)>>>(deviceData, deviceAssignments, deviceCentroids, deviceClusterCounts, numPoints, dimensions, clusters);
        hipDeviceSynchronize();

        vector<float> prevCentroids(centroids);
        hipMemcpy(centroids.data(), deviceCentroids, clusters * dimensions * sizeof(float), hipMemcpyDeviceToHost);
        float maxChange = 0.0f;
        for (int i = 0; i < clusters * dimensions; i++) {
            maxChange = max(maxChange, abs(centroids[i] - prevCentroids[i]));
        }
        converged = (maxChange < THRESHOLD);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "Converged at iteration " << iteration << "." << endl;
    cout << "Total execution time: " << elapsedTime << " ms" << endl;

    hipFree(deviceData);
    hipFree(deviceCentroids);
    hipFree(deviceAssignments);
    hipFree(deviceClusterCounts);

    return 0;
}
